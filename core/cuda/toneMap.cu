#include "hip/hip_runtime.h"
#include "../include/Renderer.h"

inline __device__ float4 sqrt(float4 f)
{
    return make_float4(sqrtf(f.x),
                       sqrtf(f.y),
                       sqrtf(f.z),
                       sqrtf(f.w));
}
inline __device__ float  clampf(float f) { return min(1.f,max(0.f,f)); }
inline __device__ float4 clamp(float4 f)
{
    return make_float4(clampf(f.x),
                       clampf(f.y),
                       clampf(f.z),
                       clampf(f.w));
}
  
  /*! runs a cuda kernel that performs gamma correction and float4-to-rgba conversion */
__global__ void computeFinalPixelColorsKernel(uint32_t *finalColorBuffer,
                                                float4   *denoisedBuffer,
                                                vec2i     size)
{
    int pixelX = threadIdx.x + blockIdx.x*blockDim.x;
    int pixelY = threadIdx.y + blockIdx.y*blockDim.y;
    if (pixelX >= size.x) return;
    if (pixelY >= size.y) return;

    int pixelID = pixelX + size.x*pixelY;

    float4 f4 = denoisedBuffer[pixelID];
    f4 = clamp(sqrt(f4));
    uint32_t rgba = 0;
    rgba |= (uint32_t)(f4.x * 255.9f) <<  0;
    rgba |= (uint32_t)(f4.y * 255.9f) <<  8;
    rgba |= (uint32_t)(f4.z * 255.9f) << 16;
    rgba |= (uint32_t)255             << 24;
    finalColorBuffer[pixelID] = rgba;
}

void Renderer::ComputeFinalPixelColors()
{
    vec2i fbSize = launchParams.frame.size;
    vec2i blockSize = 32;
    vec2i numBlocks = divRoundUp(fbSize,blockSize);
    computeFinalPixelColorsKernel
      <<<dim3(numBlocks.x,numBlocks.y),dim3(blockSize.x,blockSize.y)>>>
      ((uint32_t*)finalColorBuffer.d_pointer(),
       (float4*)denoisedBuffer.d_pointer(),
       fbSize);
}
  
